
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define N 2048 * 20480

__global__ void cuda_vector_add(float *res, float *a, float *b, int len) {
 for (int i =0 ; i < len; i++ ) {
   res[i] = a[i] + b[i];
 }
}

int main() {
  float *res, *res_gpu, *a, *a_gpu, *b, *b_gpu;
  res = (float *)malloc(sizeof(float) * N);
  a = (float *)malloc(sizeof(float) * N);
  b = (float *)malloc(sizeof(float) * N);
  hipMalloc((void**)&res_gpu, sizeof(float) * N);
  hipMalloc((void**)&a_gpu, sizeof(float) * N);
  hipMalloc((void**)&b_gpu, sizeof(float) * N);
  for (int i = 0; i < N; i++) {
   a[i] = i;
   b[i] = i + 1;
  }
  struct timeval start, end;
  gettimeofday(&start, NULL);
  hipMemcpy(a_gpu, a, sizeof(float) * N, hipMemcpyHostToDevice);
  hipMemcpy(b_gpu, b, sizeof(float) * N, hipMemcpyHostToDevice);
  gettimeofday(&end, NULL);
  long elapsed_ms = (end.tv_sec - start.tv_sec) * 1000L; // Convert seconds to milliseconds
  elapsed_ms += (end.tv_usec - start.tv_usec) / 1000L;
  printf("execution time for data copy : %ld ms \n", elapsed_ms);
  cuda_vector_add<<<20480, 20480>>>(res_gpu, a_gpu, b_gpu, N);
  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  elapsed_ms = (end.tv_sec - start.tv_sec) * 1000L; // Convert seconds to milliseconds
  elapsed_ms += (end.tv_usec - start.tv_usec) / 1000L;
  printf("execution time to kernel execution complete: %ld ms \n", elapsed_ms);
  hipMemcpy(res, res_gpu, sizeof(float) * N, hipMemcpyDeviceToHost);
  gettimeofday(&end, NULL);
  elapsed_ms = (end.tv_sec - start.tv_sec) * 1000L; // Convert seconds to milliseconds
  elapsed_ms += (end.tv_usec - start.tv_usec) / 1000L;
  printf("execution time : %ld ms \n", elapsed_ms);
  return 0;
}
